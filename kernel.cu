
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 512
#define SIZE 4096
__global__ void histo_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
	
	
	  /*************************************************************************/
	int threadId = threadIdx.x;
	int blockId = blockIdx.x;
	int p = (blockId * blockDim.x) + threadId;
	__shared__ unsigned int histo_array[SIZE];

	if(num_bins > BLOCK_SIZE) {
		for(int i = threadId; i < num_bins; i+=BLOCK_SIZE){
			if(i < num_bins){
				histo_array[i] = 0;
					}			
			}
		}
	else{
		if(threadId < num_bins){
		histo_array[threadId] = 0;
	}
	} 
	
	__syncthreads();

	if(p < num_elements) {
		atomicAdd(&(histo_array[input[p]]),1);
 
	}
	__syncthreads();

	if(num_bins > BLOCK_SIZE) {
                for(int i = threadId; i < num_bins; i+=BLOCK_SIZE){
                        if(i < num_bins){
                                atomicAdd(&(bins[i]),histo_array[i]);
                                        }
                        }
                }
        else{
                if(threadId < num_bins){
                atomicAdd(&(bins[threadId]),histo_array[threadId]);
        }
        }

        

}

void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

	  /*************************************************************************/
    //INSERT CODE HERE


	  /*************************************************************************/
	dim3 threadPerBlock(BLOCK_SIZE,1, 1);
	dim3 blockPerGrid(ceil(num_elements/(float)BLOCK_SIZE),1,1);
	histo_kernel<<<blockPerGrid,threadPerBlock>>>(input, bins, num_elements, num_bins);
}


